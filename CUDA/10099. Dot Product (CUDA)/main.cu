#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdint.h>
#define MAXN 16777216
#define BlockSize 512
 
__device__ uint32_t rotate_left(uint32_t x, uint32_t n) {
    return  (x << n) | (x >> (32-n));
}
__device__ uint32_t encrypt(uint32_t m, uint32_t key) {
    return (rotate_left(m, key&31) + key)^key;
}
 
__global__ void vecDot(uint32_t int_array[MAXN/BlockSize], uint32_t key1, uint32_t key2, int N)
{
    uint32_t sum = 0;
    int start = blockIdx.x * BlockSize;
    for (int i = start; i < N && i < start + BlockSize; i++)
        sum += encrypt(i, key1) * encrypt(i, key2);
 
    int_array[blockIdx.x] = sum;
}
 
uint32_t A[MAXN/BlockSize], B[MAXN/BlockSize], C[MAXN/BlockSize];
int main(int argc, char *argv[]) {
    int N;
    uint32_t *device_array;
    uint32_t key1, key2;
    hipMalloc((void **)&device_array, (MAXN / BlockSize) * sizeof(uint32_t));
 
    while (scanf("%d %" PRIu32 " %" PRIu32, &N, &key1, &key2) == 3) {
        hipMemcpy(device_array, A, (MAXN / BlockSize) * sizeof(uint32_t), hipMemcpyHostToDevice);
 
        int total_group = N / BlockSize + 1;
        if (total_group % BlockSize != 0)
            total_group = (total_group/BlockSize + 1) * BlockSize;
 
        vecDot <<< total_group, 1 >>> (device_array, key1, key2, N);
 
        hipMemcpy(C, device_array, (MAXN / BlockSize) * sizeof(uint32_t), hipMemcpyDeviceToHost);
 
        uint32_t sum = 0;
        for (int i = 0; i < total_group; i++)
            sum += C[i];
        printf("%" PRIu32 "\n", sum);
    }
    return 0;
}