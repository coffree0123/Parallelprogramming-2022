/* header */
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAXN 2022
#define BLOCK 16
 
/* print */
void print(char D[MAXN][MAXN], int n) {
    for (int i = 1; i <= n; i++) {
        for (int j = 1; j <= n; j++) printf("%d", D[i][j]);
        printf("\n");
    }
}
 
__global__ void game(int N, char A[MAXN][MAXN], char B[MAXN][MAXN]) {
    int nln = 0;
    int global_row = blockIdx.x * BLOCK + threadIdx.x + 1;
    int global_col = blockIdx.y * BLOCK + threadIdx.y + 1;
 
    int d[8][2] = {{-1, -1}, {-1, 0}, {0, -1}, {-1, 1},
                    {1, -1},  {0, 1},  {1, 0},  {1, 1}};
 
    if(global_row >= (N + 1) || global_col >= (N + 1)) return;
 
    for (int k = 0; k < 8; k++) {
        nln += A[global_row + d[k][0]][global_col + d[k][1]];
    }
    B[global_row][global_col] = ((A[global_row][global_col] == 0 && nln == 3) || (A[global_row][global_col] == 1 && (nln == 2 || nln == 3)));
 
}
 
char A[MAXN][MAXN];
char B[MAXN][MAXN];
 
/* main */
int main() {
    int N, generation;
 
    char *device_A;
 
    char *device_B;
 
    hipMalloc((void **)&device_A, MAXN * MAXN * sizeof(char));
 
    hipMalloc((void **)&device_B, MAXN * MAXN * sizeof(char));
 
    scanf("%d %d", &N, &generation);
 
    char str[MAXN];
    for (int i = 1; i <= N; i++) {
        scanf("%s", str);
        for (int j = 1; j <= N; j++) {
            A[i][j] = (str[j - 1] == '1')? 1 :0;
        }
    }
 
    hipMemcpy(device_A, A, MAXN * MAXN * sizeof(char), hipMemcpyHostToDevice);
 
    hipMemcpy(device_B, B, MAXN * MAXN * sizeof(char), hipMemcpyHostToDevice);
 
    int fake_N = ((N + 1 + BLOCK) / BLOCK);
    dim3 grid(fake_N, fake_N);
    dim3 block(BLOCK, BLOCK);
 
    for (int g = 0; g < generation; g++) {
        // printf("%d\n\n", g+1);
        if (g % 2 == 0) {
            // From A to B
            game <<< grid, block >>> (N, (char (*)[MAXN])device_A, (char (*)[MAXN])device_B);
            // cudaMemcpy(B, device_B, MAXN * MAXN * sizeof(char), cudaMemcpyDeviceToHost);
            // print(B, N);
        } else {
            // From B to A
            game <<< grid, block >>> (N, (char (*)[MAXN])device_B, (char (*)[MAXN])device_A);
            // cudaMemcpy(A, device_A, MAXN * MAXN * sizeof(char), cudaMemcpyDeviceToHost);
            // print(A, N);
        }
    }
 
    /* printcell */
    if (generation % 2 == 0) {
        hipMemcpy(A, device_A, MAXN * MAXN * sizeof(char), hipMemcpyDeviceToHost);
        print(A, N);
    } else {
        hipMemcpy(B, device_B, MAXN * MAXN * sizeof(char), hipMemcpyDeviceToHost);
        print(B, N);
    }
}
/* end */